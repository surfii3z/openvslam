#include "hip/hip_runtime.h"
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include <hip/hip_runtime.h>
#include "openvslam/cuda/mat_norm.hpp"
#include <hip/hip_runtime_api.h>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaarithm.hpp>
// #include <Utils.hpp>
#include "openvslam/cuda/Allocator.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cuda::device;

namespace openvslam { namespace cuda {

__device__ uint8_t d_val;

MatNormGPU::MatNormGPU() {
    // GpuMat should be already stored in memory
    //hipMallocManaged(&subtract_val, sizeof(uint8_t));
}

MatNormGPU::~MatNormGPU() {
}

__global__
void kernel_get_mat_pixel (uint8_t * src, int w, int step) //uint8_t * pSub)
{
    // assuming that we resized it to CV_32F so the channel number is 1
    //*d_subtract_val = src[ (w*step) + (w)];
    //*pSub = src[ (w * step) + (w)];
    d_val = src[ (w * step) + w];
}

void MatNormGPU::setSubtractValue(const cv::cuda::GpuMat _img, int w)
{
    uint8_t subtract_val;
    //hipMalloc(&d_subtract_val, sizeof(int));
    //uint8_t * imgPtr;
    //hipMalloc((void **)&imgPtr, _img.rows*_img.step);
    //hipMemcpy(imgPtr, _img.ptr<uint8_t>(), _img.rows*_img.step, hipMemcpyDeviceToDevice);
    //std::cout << "start subtract value" << std::endl;
    //hipMallocManaged(&subtract_val, sizeof(uint8_t));

    //SET_CLOCK(t0);
    kernel_get_mat_pixel<<<1, 1>>>(_img.data, w, _img.step);
    //SET_CLOCK(t1);
    //cout << TIME_DIFF(t1, t0) << endl;
    //std::cout << "finish kernel command" << std::endl;

    hipMemcpyFromSymbol(&subtract_val, HIP_SYMBOL(d_val), sizeof(uint8_t), 0, hipMemcpyDeviceToHost);
    subMat = cv::cuda::GpuMat(_img.rows, _img.cols, _img.type(), subtract_val, cuda::gpu_mat_allocator);
    //hipMemcpy(subtract_val, d_subtract_val, sizeof(int), hipMemcpyDeviceToHost);
    //hipFree(d_subtract_val);
    //std::cout << "gpu subtract value " << subtract_val << " here" << std::endl;
}

__global__
void kernel_subtract_pixel_from_mat (uint8_t * src, int MaxRows, int MaxCols, int step, int sub)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x; //Row number
    int rowStride = blockDim.x * gridDim.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y; //Column number
    int colStride = blockDim.y * gridDim.y;

    //unsigned int ch = blockIdx.z * blockDim.z + threadIdx.z; //Channel 0
    for (int i = row; i < MaxRows; i += rowStride) {
	for (int j = col; j < MaxCols; j += colStride) {

    	    if (row<MaxRows && col<MaxCols) {
        	int idx = i * step + j; // maxChannels is 1 and ch is 0
        	src[idx] = src[idx] - sub;
    	    }
    	}
    }
}

void MatNormGPU::subtract_pixel_from_mat (cv::cuda::GpuMat _img)
{
    //std::cout << "start subtract pixel " << int(subtract_val) << std::endl;

    //const dim3 block(16, 16);
    //const dim3 grid(((_img.cols + block.x - 1) / block.x), ((_img.rows + block.y - 1)/ block.y));

    //kernel_subtract_pixel_from_mat<<<grid, block>>> (_img.data,_img.rows, _img.cols, _img.step, subtract_val);
    //cv::cuda::subtract(_img, Scalar::all(subtract_val), _img);
    cv::cuda::subtract(_img, subMat, _img);
    //std::cout << "finish subtract pixel" << std::endl;
}

} }
