
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include <hip/hip_runtime_api.h>
#include "openvslam/cuda/stereo_matches.hpp"
// #include <Utils.hpp>

using namespace cv;
using namespace cv::cuda;
using namespace cv::cuda::device;

namespace openvslam { namespace cuda {

  StereoMatching::StereoMatching(int w) : maxKeypoints(maxKeypoints), mDescriptors(maxKeypoints, 32, CV_8UC1), mDescriptorsRight(maxKeypoints, 32, CV_8UC1) {
    checkCudaErrors( hipStreamCreate(&stream) );
    cvStream = StreamAccessor::wrapStream(stream);
    checkCudaErrors( hipMalloc(&mvKeys, sizeof(KeyPoint) * maxKeypoints) );
    checkCudaErrors( hipMalloc(&mvKeysRight, sizeof(KeyPoint) * maxKeypoints) );
  }

  StereoMatching::~StereoMatching() {
    cvStream.~Stream();
    checkCudaErrors( hipFree(mvKeys) );
    checkCudaErrors( hipFree(mvKeysRight) );
    checkCudaErrors( hipStreamDestroy(stream) );
  }



} }
