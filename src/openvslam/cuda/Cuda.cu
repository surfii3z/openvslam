#include <hip/hip_runtime_api.h>
#include "openvslam/cuda/Cuda.hpp"
namespace openvslam { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
